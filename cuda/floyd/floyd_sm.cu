#include "hip/hip_runtime.h"
// includes, system
#include <algorithm>

// includes CUDA
#include <hip/hip_runtime.h>

#include "floyd.h"


#define MAX_MAT_SIZE 4096
#define MAX_BLOCK_SIZE 1024

using namespace std;

__device__ inline void update_distance(const int size_mat, const int i, 
                        const int j, const int k, int *mat_global,
                        int row_shared[], int col_shared[]) {
    int i0 = i * size_mat + j;
    // int i1 = i * size_mat + k;
    // int i2 = k * size_mat + j;
    if (col_shared[i] != -1 && row_shared[j] != -1) {
        int sum = (col_shared[i] + row_shared[j]);
        if (mat_global[i0] == -1 || sum < mat_global[i0]) mat_global[i0] = sum;
    }
}


__global__ void update_mat_on_k(const int size_mat, const int k, int *mat_global) {
    __shared__ int row_shared[MAX_MAT_SIZE];
    __shared__ int col_shared[MAX_MAT_SIZE];

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index % size_mat;
    const int j = index / size_mat;
    if (j == 0) {
        row_shared[threadIdx.x] = mat_global[k * size_mat + i];
        col_shared[threadIdx.x] = mat_global[i * size_mat + k];
    }
    __syncthreads();
    update_distance(size_mat, i, j, k, mat_global, row_shared, col_shared);
    __syncthreads();
}


void PL_APSP(int *mat, const size_t size_mat) {
    int *mat_global;

    int num_node = size_mat * size_mat;
    int block_size = min(size_mat, (size_t) MAX_BLOCK_SIZE);
    int num_block = num_node / block_size;

    hipMalloc(&mat_global, sizeof(int) * num_node);
    hipMemcpy(mat_global, mat, sizeof(int) * num_node, hipMemcpyHostToDevice);

    dim3 dimGrid(num_block, 1, 1);
    dim3 dimBlock(block_size, 1, 1);

    for (int k = 0; k < size_mat; k++) {
        update_mat_on_k<<<dimGrid, dimBlock>>>(size_mat, k, mat_global);
    }

    hipMemcpy(mat, mat_global, sizeof(int) * num_node, hipMemcpyDeviceToHost);
}
